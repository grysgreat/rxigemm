#include "hip/hip_runtime.h"
// 这是程序二的模板程序，我们已经准备好了加载数据集和加载程序一模型参数的部分，请实现CUDA的深度学习推理过程，请严格保持输出格式输出
// 编译的命令为：nvcc -arch=sm_70 --cudart shared test.cu -o test
#include <chrono>
#include <cmath>
#include <cstring>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <string>
#include <vector>
#include <random>
#include <cstdint>
#define PROF_CUDA_EVENT

#ifdef TIMEPROF
#include "timeprof.h"
#else
#define timeprof_start_(x)
#define timeprof_end_()
#define timeprof_print_frame_sorted_()
#endif

// 打印矩阵
void printMatrix(float matrix[], int rows, int cols) {
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            // 计算数组中元素的索引
            int index = i * cols + j;
            printf("%.4f   \t", matrix[index]);
        }
        std::cout << std::endl;
    }
}

// 高精度打印矩阵
void printMatrix_h(float matrix[], int rows, int cols) {
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            // 计算数组中元素的索引
            int index = i * cols + j;
            printf("%.8f   \t", matrix[index]);
        }
        std::cout << std::endl;
    }
}

// 打印int矩阵
void printMatrix_int(int matrix[], int rows, int cols) {
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            // 计算数组中元素的索引
            int index = i * cols + j;
            printf("%d\t\t", matrix[index]);
        }
        std::cout << std::endl;
    }
}

//矩阵拷贝函数
template <typename T>
void xcopy(const T matrix1[],T matrix2[], int rows, int cols ) {
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
			matrix2[i*cols+j] = matrix1[i*cols+j];
        }
    }
}

// 定义矩阵转置函数
template <typename T>
void xtrans(T matrix[],T result[] , int rows, int cols) {

	T tmp[rows*cols];
    // 执行矩阵转置   
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            // 计算原矩阵中元素的索引
            int originalIndex = i * cols + j;
            // 计算转置后矩阵中元素的索引
            int transposedIndex = j * rows + i;
            // 进行转置
            tmp[transposedIndex] = matrix[originalIndex];
        }
    }
    xcopy<T>(tmp,result,cols,rows);
}


template <typename Ti,typename To>
void quantitize(Ti* matrix_in,To* matrix_out,int rows,int cols,float lambda,char type){

    if(type == 'q'){
        for(int i=0;i<rows;i++){
            for(int j=0;j<cols;j++){
                matrix_out[i*cols+j] = matrix_in[i*cols+j]*lambda;
            }
        }
    }
    else if(type == 'd'){
        float l1 = 1.0/lambda;
        for(int i=0;i<rows;i++){
            for(int j=0;j<cols;j++){
                matrix_out[i*cols+j] = (To)matrix_in[i*cols+j]*l1;
            }
        }        
    }

    // int max_exp;
    // std::frexp(max, &max_exp);
}




template <typename T>
T get_max(T* matrix,int rows,int cols){
    T maxM=0;
    for(int i=0;i<rows;i++){
        for(int j=0;j<cols;j++){
            maxM = max(std::abs(maxM),std::abs(matrix[i*cols+j]));
        }
    }
    return maxM;
}


template <typename T>
T get_min(T* matrix,int rows,int cols){
    T minM=1000000;
    for(int i=0;i<rows;i++){
        for(int j=0;j<cols;j++){
            if(std::abs(matrix[i*cols+j])>1e-20)
                minM = min(std::abs(minM),std::abs(matrix[i*cols+j]));
        }
    }
    return minM;
}


template <typename T>
void get_min_vec(T* matrix,T* vec, int rows,int cols,char type){
    
    if(type == 'r'){
        for(int i=0;i<rows;i++){
            T minM=2553555;
            for(int j=0;j<cols;j++){
                minM = min(std::abs(minM),std::abs(matrix[i*cols+j]));
            }
            vec[i] = minM;
        }
    }
    if(type == 'c'){
        for(int j=0;j<cols;j++){
            T minM=2553555;
            for(int i=0;i<rows;i++){
                minM = min(std::abs(minM),std::abs(matrix[i*cols+j]));
            }
            vec[j] = minM;
        }
    }    


}

template <typename T>
void get_max_vec(T* matrix,T* vec, int rows,int cols,char type){
    
    if(type == 'r'){
        for(int i=0;i<rows;i++){
            T minM=0;
            for(int j=0;j<cols;j++){
                minM = max(std::abs(minM),std::abs(matrix[i*cols+j]));
            }
            vec[i] = minM;
        }
    }
    if(type == 'c'){
        for(int j=0;j<cols;j++){
            T minM=0;
            for(int i=0;i<rows;i++){
                minM = max(std::abs(minM),std::abs(matrix[i*cols+j]));
            }
            vec[j] = minM;
        }
    }    


}


template <typename T>
void get_avg_vec(T* matrix,T* vec, int rows,int cols,char type){
    
    if(type == 'r'){
        for(int i=0;i<rows;i++){
            T sum=0;
            for(int j=0;j<cols;j++){
                sum += std::abs(matrix[i*cols+j]);
            }
            vec[i] = sum/cols;
        }
    }
    if(type == 'c'){
        for(int j=0;j<cols;j++){
            T sum=0;
            for(int i=0;i<rows;i++){
                sum += std::abs(matrix[i*cols+j]);
            }
            vec[j] = sum/rows;
        }
    }    


}

template <typename T>
void xmadd(T* matrixA,T* matrixB,T* matrixC,int rows,int cols){
    for(int i=0;i<rows;i++){
        for(int j=0;j<cols;j++){
            matrixC[i*cols+j] = matrixA[i*cols+j] + matrixB[i*cols+j];
        }
    }
}

template <typename T>
// 定义矩阵乘矩阵函数
void xgemm(const T A[], const T B[], T C[], int rowsA, int colsA, int rowsB, int colsB) {

    // 确保可以进行矩阵乘法的尺寸
    if (colsA != rowsB) {
        std::cerr << "无法进行矩阵乘法，尺寸不匹配。" << std::endl;
        return;
    }
    T tmp[rowsA*colsB];
    for (int i = 0; i < rowsA; ++i) {
        for (int j = 0; j < colsB; ++j) {
            // 初始化结果矩阵中的元素为0
            tmp[i * colsB + j] = 0;
            for (int k = 0; k < colsA; ++k) {
                // 矩阵乘法的累加步骤
                tmp[i * colsB + j] += A[i * colsA + k] * B[k * colsB + j];
            }
        }
    }
    
    for (int i = 0; i < rowsA; ++i) {
        for (int j = 0; j < colsB; ++j) {
            C[i * colsB + j]  = tmp[i * colsB + j];
		}
	}
    
}
template <typename T>
void xgemm_nt(const T A[], const T B[], T C[],const int rowsA,const int colsA,const int rowsB,const int colsB) {

    // 确保可以进行矩阵乘法的尺寸
    if (colsA != rowsB) {
        std::cerr << "无法进行矩阵乘法，尺寸不匹配。" << std::endl;
        return;
    }
    T tmp[rowsA*colsB];
    for (int i = 0; i < rowsA; ++i) {
        for (int j = 0; j < colsB; ++j) {
            // 初始化结果矩阵中的元素为0
            tmp[j * rowsA + i] = 0;
            for (int k = 0; k < colsA; ++k) {
                // 矩阵乘法的累加步骤
                tmp[j * rowsA + i] += A[i * colsA + k] * B[j * colsA + k];
            }
        }
    }
    
    for (int i = 0; i < rowsA; ++i) {
        for (int j = 0; j < colsB; ++j) {
            C[i * colsB + j]  = tmp[i * colsB + j];
		}
	}
    
}

void xgemm_nti(const int8_t* A, const int8_t* B, int32_t* C, int rowsA, int colsA, int rowsB, int colsB) {

    // 确保可以进行矩阵乘法的尺寸
    if (colsA != rowsB) {
        std::cerr << "无法进行矩阵乘法，尺寸不匹配。" << std::endl;
        return;
    }
    int32_t *tmp = (int32_t *)malloc(sizeof(int32_t) * rowsA*colsB);
    for (int i = 0; i < rowsA; ++i) {
        for (int j = 0; j < colsB; ++j) {
            // 初始化结果矩阵中的元素为0
            tmp[j * rowsA + i] = 0;
            for (int k = 0; k < colsA; ++k) {
                // 矩阵乘法的累加步骤
                tmp[j * rowsA + i] += static_cast<int32_t>(A[i * colsA + k]) * static_cast<int32_t>(B[j * colsA + k]);
            }
        }
    }
    
    for (int i = 0; i < rowsA; ++i) {
        for (int j = 0; j < colsB; ++j) {
            C[i * colsB + j]  = tmp[i * colsB + j];
		}
	}
    
}


template <typename T>
int get_nnz(T *denseMatrix, int numRows,int numCols){
    int nnz = 0; // Number of non-zero elements
    for (int i = 0; i < numRows; ++i) {
        for (int j = 0; j < numCols; ++j) {
            if (denseMatrix[i*numCols+j] != 0) {
                ++nnz;
            }
        }
    }
    return nnz;
}


// Function to convert a dense matrix to CSR format
template <typename T>
void denseToCSR(T* denseMatrix,T *values,int *colIndex,int *rowPtr, int numRows, int numCols) {
    int nnz = 0;
    for (int i = 0; i < numRows; ++i) {
        rowPtr[i] = nnz;
        for (int j = 0; j < numCols; ++j) {
            if (denseMatrix[i*numCols+j] != 0) {
                values[nnz] = denseMatrix[i*numCols+j];
                colIndex[nnz] = j;
                ++nnz;
            }
        }
    }
    rowPtr[numRows] = nnz;

}


// Function to perform matrix-vector multiplication (CSR format) spmm 
template <typename T>
void sspmm(T *values,int *colIndex,int *rowPtr, T* B, T* C,int rowsA,int colsA,int rowsB, int colsB) {

    for (int i = 0; i < rowsA; ++i) {
        for (int j = 0; j < colsB; ++j) {
            C[i*colsB+j] = 0.0;
            for (int k = rowPtr[i]; k < rowPtr[i + 1]; ++k) {
                C[i*colsB+j] += values[k] * B[colIndex[k]*colsB+j];
            }
        }
    }
}

template <typename T>
void sspmm_nt(T *values,int *colIndex,int *rowPtr, T* B, T* C,int rowsA,int colsA,int rowsB, int colsB) {

    for (int i = 0; i < rowsA; ++i) {
        for (int j = 0; j < colsB; ++j) {
            C[i+j*rowsA] = 0.0;
            for (int k = rowPtr[i]; k < rowPtr[i + 1]; ++k) {
                
                C[i+j*rowsA] += values[k] * B[colIndex[k]+j*colsA];
            }
        }
    }
}


void sspmm_nti(int8_t *values,int *colIndex,int *rowPtr, int8_t* B, int* C,int rowsA,int colsA,int rowsB, int colsB) {

    for (int i = 0; i < rowsA; ++i) {
        for (int j = 0; j < colsB; ++j) {
            C[i+j*rowsA] = 0;
            for (int k = rowPtr[i]; k < rowPtr[i + 1]; ++k) {
                
                C[i+j*rowsA] += static_cast<int32_t>(values[k]) * static_cast<int32_t>(B[colIndex[k]+j*colsA]);
            }
        }
    }
}


template <typename T>
void get_R(T matrix_in[],T matrix_cmp[],T matrixR[],int rows,int cols){
    for(int i=0;i<rows;i++){
        for(int j=0;j<cols;j++){
            matrixR[i*cols+j] = (matrix_in[i*cols+j] - matrix_cmp[i*cols+j]);
        }
    }
}

template <typename T>
void get_error(T matrix_ref[],T matrix_cmp[],T matrixR[],int rows,int cols){
    for(int i=0;i<rows;i++){
        for(int j=0;j<cols;j++){
            matrixR[i*cols+j] = std::abs(matrix_ref[i*cols+j] - matrix_cmp[i*cols+j])/std::abs(matrix_ref[i*cols+j]);
        }
    }
}

template <typename T>
T get_Ferror(T matrix_ref[],T matrix_cmp[],int rows,int cols){

    T sumR=0,sum=0;
    for(int i=0;i<rows;i++){
        for(int j=0;j<cols;j++){
            sumR+=(matrix_ref[i*cols+j] - matrix_cmp[i*cols+j])*(matrix_ref[i*cols+j] - matrix_cmp[i*cols+j]);
            sum+=(matrix_ref[i*cols+j])*(matrix_ref[i*cols+j]);
        }
    }

    T ans = sqrt(sumR)/sqrt(sum);
    return ans;

}


template <typename T>
void reduce_Residual(T* matrix,T* matrixR,int rows,int cols,int N,T max,int threshold){
    int max_exp;
    std::frexp(max, &max_exp);    
    int judge = max_exp - N + threshold;

    for(int i=0;i<rows;i++){
        for(int j=0;j<cols;j++){
            int aij_exp;
            std::frexp(matrix[i*cols+j], &aij_exp); 
            if(aij_exp>judge){
                matrixR[i*cols+j] = 0;
            }
        }
    }        
}



template <typename T>
void reduce_Matrix(T* matrix,T* vec,int rows,int cols,T Mlam_k,char type){
    float cnt=0;
    if(type == 'r'){
        for(int i=0;i<rows;i++){
            T judge = Mlam_k*vec[i];
            for(int j=0;j<cols;j++){
                if(std::abs(matrix[i*cols+j])<judge) {
                    matrix[i*cols+j]=0;
                    cnt++;
                }
            }
        }        
    }
    if(type == 'c'){
        for(int j=0;j<cols;j++){
            vec[j]*=Mlam_k;
        }        
        for(int i=0;i<rows;i++){
            for(int j=0;j<cols;j++){
                // T judge = Mlam_k*vec[j];
                if(std::abs(matrix[i*cols+j])<vec[j]) {
                    matrix[i*cols+j]=0;
                    cnt++;
                }
            }
        }        
    }
    float spasity = 1-(cnt/(float)(rows*cols));
    printf("%f\n",spasity);
}


template <typename T>
void generate_matrix(T* matrix,int rows,int cols,char type ){
    // 创建一个随机数引擎
    std::random_device rd;
    std::mt19937 gen(rd());
    // 创建一个均匀分布，范围是[0, 1)
    std::uniform_real_distribution<float> dis(0.0, 1.0);
    //std::normal_real_distribution<float> dis(0.0, 1.0);

    if(type == 'u'){
        for(int i=0;i<rows;i++){
            for(int j=0;j<cols;j++){
                matrix[i*cols+j] = dis(gen);
                if(i==j) matrix[i*cols+j] = (matrix[i*cols+j]);
                else  matrix[i*cols+j]=(matrix[i*cols+j])/=100;
            }
        }        
    }
    else{
        for(int i=0;i<rows;i++){
            for(int j=0;j<cols;j++){
                T U1 = rand() * 1.0f / RAND_MAX; // 0~1均匀分布
                T U2 = rand() * 1.0f / RAND_MAX; // 0~1均匀分布
                T Z = sqrt(-2 * log(U1))*cos(2 * M_PI * U2);// 均值为0，方差为1的正态分布
                T Y = 1 +  Z; // 均值为1，方差为4的正态分布
                matrix[i*cols+j] = Y;
            }
        }        
    }
}
template <typename T>
void splitMatrix(T A[], T B[],  int rows, int cols, T judge){
    for(int i=0;i<rows;i++){
        for(int j=0;j<cols;j++){
            if(abs(A[i*cols+j])<judge){
                B[i*cols+j] = A[i*cols+j];
                A[i*cols+j] = 0;
            } else {
                B[i*cols+j] = 0;
            }
        }
    }        
}

template <typename T>
T get_sparsity(T A[],int rows, int cols){
    int cnt=0;
    for(int i=0;i<rows;i++){
        for(int j=0;j<cols;j++){
            if(A[i*cols+j]!=0){
                cnt++;
            } 
        }
    }     
    T sp = (T)cnt / (T)(rows*cols);
    return sp;
}


// 定义矩阵乘矩阵函数---- 残差法
template <typename T,int digit>
void xigemm(T A[], T B[], T C[], int rowsA, int colsA, int rowsB, int colsB,float threadhoud,char type) {

    // 确保可以进行矩阵乘法的尺寸
    if (colsA != rowsB) {
        std::cerr << "无法进行矩阵乘法，尺寸不匹配。" << std::endl;
        return;
    }
    int8_t *A_int8 = (int8_t *)malloc(sizeof(int8_t) * rowsA*colsA);
    int8_t *B_int8 = (int8_t *)malloc(sizeof(int8_t) * rowsB*colsB);
    int32_t *C_int32 = (int32_t *)malloc(sizeof(int32_t) * rowsA*colsB);

    int *A_int = (int *)malloc(sizeof(int) * rowsA*colsA);
    int *B_int = (int *)malloc(sizeof(int) * rowsB*colsB);
    int *C_int = (int *)malloc(sizeof(int) * rowsA*colsB);
    int *AR_int = (int *)malloc(sizeof(int) * rowsA*colsA);
    int *BR_int = (int *)malloc(sizeof(int) * rowsB*colsB);

    T *C_copy = (T *)malloc(sizeof(T) * rowsA * colsB);
    T *C_buffer = (T *)malloc(sizeof(T) * rowsA * colsB);
    T *A_p = (T *)malloc(sizeof(T) * rowsA * colsA);
    T *B_p = (T *)malloc(sizeof(T) * rowsB * colsB);
    T *C_rows = (T *)malloc(sizeof(T) * rowsA * colsB);
    T *C_cols = (T *)malloc(sizeof(T) * rowsA * colsB);
    T *A_copy = (T *)malloc(sizeof(T) * rowsA * colsA);
    T *B_copy = (T *)malloc(sizeof(T) * rowsB * colsB);

   

    xcopy<T>(A,A_copy, rowsA, colsA);
    xcopy<T>(B,B_copy, rowsB, colsB);


    const int max_int = 1<<(digit-1) - 1;
    T max_mA =get_max<T>(A_copy,rowsA, colsA);
    T max_mB =get_max<T>(B,rowsB, colsB);
    T lambdaA = (T)max_int/max_mA;
    T lambdaB = (T)max_int/max_mB;

    //对A,B矩阵进行直接量化
    quantitize<T,int>(A,A_int,rowsA, colsA,lambdaA,'q');
    quantitize<T,int>(B,B_int,rowsB, colsB,lambdaB,'q');


    //计算float和int矩阵乘法得到结果矩阵
    xgemm_nt<int>(A_int,B_int,C_int,rowsA,colsA,rowsB,colsB);

    //xgemm_nti(A_int8,B_int8,C_int32,rowsA,colsA,rowsB,colsB);

    //对结果矩阵C_INT进行反量化得到C'
    T lambdaC = lambdaA*lambdaB;
    quantitize<int,T>(C_int,C_buffer,rowsA,colsB,lambdaC,'d');



    if(type == 'c' || type == 'x'){
        
        //对量化后的int矩阵反量化得到A',B'
        quantitize<int,T>(A_int,A_p,rowsA,colsA,lambdaA,'d');
        quantitize<int,T>(B_int,B_p,rowsB,colsB,lambdaB,'d');

        //计算full size的残差矩阵
        get_R<T>(A,A_p,A_p,rowsA,colsA);
        get_R<T>(B,B_p,B_p,rowsB,colsB);
        //对残差矩阵进行量化
        T max_mAR =get_max<T>(A_p,rowsA,colsA);
        T max_mBR =get_max<T>(B_p,rowsB,colsB);
        T lambdaAR = (T)max_int/max_mAR;
        T lambdaBR = (T)max_int/max_mBR;
        //对A,B残差矩阵进行直接量化
        quantitize<T,int>(A_p,AR_int,rowsA,colsA,lambdaAR,'q');
        quantitize<T,int>(B_p,BR_int,rowsB,colsB,lambdaBR,'q');


        T lambdaAnew = lambdaA,lambdaBnew = lambdaB;
        if(type == 'x'){
            //稀疏化误差修补
            T ml_kA = threadhoud*lambdaB/colsA;
            T ml_kB = threadhoud*lambdaA/rowsB;    


            get_avg_vec<T>(C_buffer,C_rows,rowsA,colsB,'r');
            get_avg_vec<T>(C_buffer,C_cols,rowsA,colsB,'c');
            
            reduce_Matrix(A_copy,C_rows,rowsA,colsA, ml_kA ,'r');
            reduce_Matrix(B_copy,C_cols,rowsB,colsB, ml_kB ,'c');

            float  min = get_min(A_copy,rowsA,colsA);
            //printf("ml_kA = %f,min =%f,C_rows=%f\n",ml_kA,min,C_rows[0]);

        // std::cout<<"otho"<<"\n";
        // printMatrix_h(A_copy,rowsA,colsA);
        // std::cout<<"\n\n";

            //对新AB进行量化
            T max_mAnew =get_max<T>(A_copy,rowsA,colsA);
            T max_mBnew =get_max<T>(B_copy,rowsB,colsB);
            lambdaAnew = (T)max_int/max_mAnew;
            lambdaBnew = (T)max_int/max_mBnew;
            quantitize<T,int>(A_copy,A_int,rowsA,colsA,lambdaAnew,'q');
            quantitize<T,int>(B_copy,B_int,rowsB,colsB,lambdaBnew,'q');
        }




        //对int误差修复矩阵反量化得到误差修复矩阵float
        T lambdaCR1 = lambdaAnew*lambdaBR;
        T lambdaCR2 = lambdaAR*lambdaBnew;

        xgemm_nt<int>(A_int,BR_int,C_int,rowsA,colsA,rowsB,colsB);
        quantitize<int,T>(C_int,C_copy,rowsA,colsB,lambdaCR1,'d');
        //使用修复矩阵补充误差
        xmadd<float>(C_copy,C_buffer,C_buffer,rowsA,colsB);

        
        xgemm_nt<int>(AR_int,B_int,C_int,rowsA,colsA,rowsB,colsB);
        quantitize<int,T>(C_int,C_copy,rowsA,colsB,lambdaCR1,'d');
        //使用修复矩阵补充误差
        xmadd<float>(C_copy,C_buffer,C_buffer,rowsA,colsB);


    }

    xcopy<T>(C_buffer,C, rowsA, colsB);

}



// 定义矩阵乘矩阵函数---- 残差法
template <typename T,int digit>
void xigemm_int8(T A[], T B[], T C[], int rowsA, int colsA, int rowsB, int colsB,float threadhoud,char type) {

    // 确保可以进行矩阵乘法的尺寸
    if (colsA != rowsB) {
        std::cerr << "无法进行矩阵乘法，尺寸不匹配。" << std::endl;
        return;
    }
    int8_t *A_int8 = (int8_t *)malloc(sizeof(int8_t) * rowsA*colsA);
    int8_t *B_int8 = (int8_t *)malloc(sizeof(int8_t) * rowsB*colsB);
    int32_t *C_int32 = (int32_t *)malloc(sizeof(int32_t) * rowsA*colsB);
    int8_t *AR_int8 = (int8_t *)malloc(sizeof(int8_t) * rowsA*colsA);
    int8_t *BR_int8 = (int8_t *)malloc(sizeof(int8_t) * rowsB*colsB);


    // int *A_int = (int *)malloc(sizeof(int) * rowsA*colsA);
    // int *B_int = (int *)malloc(sizeof(int) * rowsB*colsB);
    // int *C_int = (int *)malloc(sizeof(int) * rowsA*colsB);
    // int *AR_int = (int *)malloc(sizeof(int) * rowsA*colsA);
    // int *BR_int = (int *)malloc(sizeof(int) * rowsB*colsB);

    T *C_copy = (T *)malloc(sizeof(T) * rowsA * colsB);
    T *C_buffer = (T *)malloc(sizeof(T) * rowsA * colsB);
    T *A_p = (T *)malloc(sizeof(T) * rowsA * colsA);
    T *B_p = (T *)malloc(sizeof(T) * rowsB * colsB);
    T *C_rows = (T *)malloc(sizeof(T) * rowsA * colsB);
    T *C_cols = (T *)malloc(sizeof(T) * rowsA * colsB);
    T *A_copy = (T *)malloc(sizeof(T) * rowsA * colsA);
    T *B_copy = (T *)malloc(sizeof(T) * rowsB * colsB);

   

    xcopy<T>(A,A_copy, rowsA, colsA);
    xcopy<T>(B,B_copy, rowsB, colsB);


    const int max_int = 1<<(digit-1) - 1;
    T max_mA =get_max<T>(A_copy,rowsA, colsA);
    T max_mB =get_max<T>(B,rowsB, colsB);
    T lambdaA = (T)max_int/max_mA;
    T lambdaB = (T)max_int/max_mB;

    //对A,B矩阵进行直接量化
    quantitize<T,int8_t>(A,A_int8,rowsA, colsA,lambdaA,'q');
    quantitize<T,int8_t>(B,B_int8,rowsB, colsB,lambdaB,'q');


    //计算float和int矩阵乘法得到结果矩阵
    // xgemm_nt<int>(A_int,B_int,C_int,rowsA,colsA,rowsB,colsB);

    xgemm_nti(A_int8,B_int8,C_int32,rowsA,colsA,rowsB,colsB);

    //对结果矩阵C_INT进行反量化得到C'
    T lambdaC = lambdaA*lambdaB;
    quantitize<int32_t,T>(C_int32,C_buffer,rowsA,colsB,lambdaC,'d');



    if(type == 'c' || type == 'x'){
        
        //对量化后的int矩阵反量化得到A',B'
        quantitize<int8_t,T>(A_int8,A_p,rowsA,colsA,lambdaA,'d');
        quantitize<int8_t,T>(B_int8,B_p,rowsB,colsB,lambdaB,'d');

        //计算full size的残差矩阵
        get_R<T>(A,A_p,A_p,rowsA,colsA);
        get_R<T>(B,B_p,B_p,rowsB,colsB);
        //对残差矩阵进行量化
        T max_mAR =get_max<T>(A_p,rowsA,colsA);
        T max_mBR =get_max<T>(B_p,rowsB,colsB);
        T lambdaAR = (T)max_int/max_mAR;
        T lambdaBR = (T)max_int/max_mBR;
        //对A,B残差矩阵进行直接量化
        quantitize<T,int8_t>(A_p,AR_int8,rowsA,colsA,lambdaAR,'q');
        quantitize<T,int8_t>(B_p,BR_int8,rowsB,colsB,lambdaBR,'q');

        //printf("rowsA = %d,colsA = %d,rowsB = %d,colsB =%d\n",rowsA,colsA,rowsB,colsB);

        T lambdaAnew = lambdaA,lambdaBnew = lambdaB;
        if(type == 'x'){
            //稀疏化误差修补
            T ml_kA = threadhoud*lambdaB/colsA;
            T ml_kB = threadhoud*lambdaA/rowsB;    


            get_avg_vec<T>(C_buffer,C_rows,rowsA,colsB,'r');
            get_avg_vec<T>(C_buffer,C_cols,rowsA,colsB,'c');
            
            reduce_Matrix(A_copy,C_rows,rowsA,colsA, ml_kA ,'r');
            reduce_Matrix(B_copy,C_cols,rowsB,colsB, ml_kB ,'c');

            //float  min = get_min(A_copy,rowsA,colsA);
            //printf("ml_kA = %f,min =%f,C_rows=%f\n",ml_kA,min,C_rows[0]);

        // std::cout<<"otho"<<"\n";
        // printMatrix_h(A_copy,rowsA,colsA);
        // std::cout<<"\n\n";

            //对新AB进行量化
            T max_mAnew =get_max<T>(A_copy,rowsA,colsA);
            T max_mBnew =get_max<T>(B_copy,rowsB,colsB);
            lambdaAnew = (T)max_int/max_mAnew;
            lambdaBnew = (T)max_int/max_mBnew;
            quantitize<T,int8_t>(A_copy,A_int8,rowsA,colsA,lambdaAnew,'q');
            quantitize<T,int8_t>(B_copy,B_int8,rowsB,colsB,lambdaBnew,'q');
        
            //对int误差修复矩阵反量化得到误差修复矩阵float
            T lambdaCR1 = lambdaAnew*lambdaBR;
            T lambdaCR2 = lambdaAR*lambdaBnew;


            int nnzA = get_nnz<int8_t>(A_int8, rowsA, colsA);
            int nnzB = get_nnz<int8_t>(B_int8, colsB, rowsB);

            float spA = (float)nnzA/((float)rowsA*(float)colsA);
            float spB = (float)nnzB/((float)rowsB*(float)colsB);
            printf("rowsA = %d,colsA = %d,rowsB = %d,colsB =%d, nnzA = %d,nnzB = %d,spA=%f,spB=%f\n",rowsA,colsA,rowsB,colsB,nnzA,nnzB,spA,spB);
            if(spA<0.1){
              int8_t* valuesA = new int8_t[nnzA];
              int* colIndexA = new int[nnzA];
              int* rowPtrA = new int[rowsA + 1];
              denseToCSR<int8_t>(A_int8, valuesA,colIndexA,rowPtrA, rowsA, colsA);
              sspmm_nti(valuesA,colIndexA,rowPtrA, BR_int8, C_int32,rowsA, colsA,rowsB, colsB);
            }else {
              xgemm_nti(A_int8,BR_int8,C_int32,rowsA,colsA,rowsB,colsB);
            }
            //
            quantitize<int32_t,T>(C_int32,C_copy,rowsA,colsB,lambdaCR1,'d');
            //使用修复矩阵补充误差
            xmadd<float>(C_copy,C_buffer,C_buffer,rowsA,colsB);

            if(spB<0.1){
              xtrans<int8_t>(B_int8,B_int8,rowsB,colsB);
              xtrans<int8_t>(AR_int8,AR_int8,rowsA, colsA);
              int8_t* valuesB = new int8_t[nnzB];
              int* colIndexB = new int[nnzB];
              int* rowPtrB = new int[colsB + 1];
              denseToCSR<int8_t>(B_int8, valuesB,colIndexB,rowPtrB, colsB, rowsB);
              sspmm_nti(valuesB,colIndexB,rowPtrB, AR_int8, C_int32,colsB, rowsB,colsA, rowsA);
              xtrans<int>(C_int32,C_int32,colsB,rowsA);

  
            }else {
              xgemm_nti(AR_int8,B_int8,C_int32,rowsA,colsA,rowsB,colsB);
            }
            quantitize<int32_t,T>(C_int32,C_copy,rowsA,colsB,lambdaCR1,'d');
            //使用修复矩阵补充误差
            xmadd<float>(C_copy,C_buffer,C_buffer,rowsA,colsB);

 
        } else {
            //对int误差修复矩阵反量化得到误差修复矩阵float
            T lambdaCR1 = lambdaAnew*lambdaBR;
            T lambdaCR2 = lambdaAR*lambdaBnew;

            xgemm_nti(A_int8,BR_int8,C_int32,rowsA,colsA,rowsB,colsB);

            quantitize<int32_t,T>(C_int32,C_copy,rowsA,colsB,lambdaCR1,'d');
            //使用修复矩阵补充误差
            xmadd<float>(C_copy,C_buffer,C_buffer,rowsA,colsB);

            xgemm_nti(AR_int8,B_int8,C_int32,rowsA,colsA,rowsB,colsB);
            quantitize<int32_t,T>(C_int32,C_copy,rowsA,colsB,lambdaCR1,'d');
            //使用修复矩阵补充误差
            xmadd<float>(C_copy,C_buffer,C_buffer,rowsA,colsB);

  
        }





    }

    xcopy<T>(C_buffer,C, rowsA, colsB);

}


// 定义矩阵乘矩阵函数---- 指数切分法
template <typename T,int digit>
void xigemm_e(T A[], T B[], T C[], int rowsA, int colsA, int rowsB, int colsB) {


    T A_copy[10000],B_copy[10000],A_0[10000],B_0[10000],A_B_0[10000],A_0_B[10000],A_B[10000],A_0_B_0[10000];

    xcopy<float>(A,A_copy,rowsA,colsA);
    xcopy<float>(B,B_copy,rowsB,colsB);
    T max_mA =get_max<T>(A,rowsA,colsA);
    T max_mB =get_max<T>(B,rowsB,colsB);    
    T min_mA =get_min<T>(A,rowsA,colsA);
    T min_mB =get_min<T>(B,rowsB,colsB);    




    T judgeA = exp((log(max_mA) + log(min_mA))/2);
    T judgeB = exp((log(max_mB) + log(min_mB))/2);


    splitMatrix<float>(A_copy,A_0,rowsA,colsA,judgeA);
    splitMatrix<float>(B_copy,B_0,rowsB,colsB,judgeB);


    // printMatrix_h(B_0,rowsB,colsB);
    // std::cout<<"\n\n";
    float sa = get_sparsity(A_0,rowsA,colsA);
    float sb = get_sparsity(B_0,rowsB,colsB);
    printf("sparsity = %f\n", sa);

    float thread_f = 1.0/(float)(1<<6);
    xigemm<T,digit>(A_copy, B_copy,A_B,rowsA,colsA,rowsB,colsB,thread_f,'o');
    xigemm<T,digit>(A_0, B_copy, A_0_B,rowsA,colsA,rowsB,colsB,thread_f,'o');
    xigemm<T,digit>(A_copy ,B_0,A_B_0,rowsA,colsA,rowsB,colsB,thread_f,'o');
    xigemm<T,digit>(A_0,B_0,A_0_B_0,rowsA,colsA,rowsB,colsB,thread_f,'o');

    // xgemm<T>(A_copy,B_copy,A_B,rowsA,colsA,rowsB,colsB);
    // xgemm<T>(A_0, B_copy, A_0_B,rowsA,colsA,rowsB,colsB);
    // xgemm<T>(A_copy, B_0, A_B_0,rowsA,colsA,rowsB,colsB);
    // xgemm<T>(A_0,B_0,A_0_B_0,rowsA,colsA,rowsB,colsB);



    xmadd<T>(A_B, A_0_B, C,rowsA,colsB);
    xmadd<T>(A_B_0,  C, C,rowsA,colsB);
    xmadd<T>(C,  A_0_B_0, C,rowsA,colsB);

}


const int in_channel = 1;
const int in_height = 28;
const int in_weight = 28;

const int batch_size = 10000;
// const int batch_size = 3;

// conv 1
const int conv_1_kernel_size = 2;
const int conv_1_stride = 2;
const int conv_1_padding = 0;

const int conv_1_out_channel = 1;
const int conv_1_out_height =
    (in_height - conv_1_kernel_size + 2 * conv_1_padding) / conv_1_stride + 1;
const int conv_1_out_width =
    (in_weight - conv_1_kernel_size + 2 * conv_1_padding) / conv_1_stride + 1;

// Linear 1
const int linear_1_in_size = conv_1_out_channel * conv_1_out_height * conv_1_out_width;
const int linear_1_out_size = 64;

// Linear 2
const int linear_2_out_size = 10;

template <typename T>
void check(T result, char const *const func, const char *const file, int const line) {
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line,
            static_cast<unsigned int>(result), hipGetErrorString(result), func);
    exit(EXIT_FAILURE);
  }
}

#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)
#define CEIL_DIV(x, y) (((x) + (y)-1) / (y))

// 读取MNIST数据集
std::vector<std::vector<float>> read_mnist_images(const std::string &path) {
  std::ifstream file(path, std::ios::binary);
  if (!file) {
    std::cout << "Cannot open file!" << std::endl;
    return {};
  }

  int magic_number = 0, num_images = 0, num_rows = 0, num_cols = 0;
  file.read((char *)&magic_number, sizeof(magic_number));
  file.read((char *)&num_images, sizeof(num_images));
  file.read((char *)&num_rows, sizeof(num_rows));
  file.read((char *)&num_cols, sizeof(num_cols));

  // Reverse Integers (MNIST data is in big endian format)
  magic_number = ((magic_number & 0xff000000) >> 24) | ((magic_number & 0x00ff0000) >> 8) |
                 ((magic_number & 0x0000ff00) << 8) | ((magic_number & 0x000000ff) << 24);
  num_images = ((num_images & 0xff000000) >> 24) | ((num_images & 0x00ff0000) >> 8) |
               ((num_images & 0x0000ff00) << 8) | ((num_images & 0x000000ff) << 24);
  num_rows = ((num_rows & 0xff000000) >> 24) | ((num_rows & 0x00ff0000) >> 8) |
             ((num_rows & 0x0000ff00) << 8) | ((num_rows & 0x000000ff) << 24);
  num_cols = ((num_cols & 0xff000000) >> 24) | ((num_cols & 0x00ff0000) >> 8) |
             ((num_cols & 0x0000ff00) << 8) | ((num_cols & 0x000000ff) << 24);

  int image_size = num_rows * num_cols;
  std::vector<std::vector<float>> images(num_images, std::vector<float>(image_size));

  // printf("num_images=%d, num_rows=%d, num_cols=%d\n", num_images, num_rows, num_cols);

  for (int i = 0; i < num_images; ++i) {
    for (int j = 0; j < image_size; ++j) {
      unsigned char pixel = 0;
      file.read((char *)&pixel, sizeof(pixel));
      images[i][j] = static_cast<float>(pixel) / 255.0f;
    }
  }

  return images;
}

// 读取MNIST label数据集
std::vector<int> read_mnist_labels(const std::string &path) {
  std::ifstream file(path, std::ios::binary);
  if (!file) {
    std::cout << "Cannot open file!" << std::endl;
    return {};
  }

  int magic_number = 0, num_items = 0;
  file.read((char *)&magic_number, sizeof(magic_number));
  file.read((char *)&num_items, sizeof(num_items));

  // Reverse Integers (MNIST data is in big endian format)
  magic_number = ((magic_number & 0xff000000) >> 24) | ((magic_number & 0x00ff0000) >> 8) |
                 ((magic_number & 0x0000ff00) << 8) | ((magic_number & 0x000000ff) << 24);
  num_items = ((num_items & 0xff000000) >> 24) | ((num_items & 0x00ff0000) >> 8) |
              ((num_items & 0x0000ff00) << 8) | ((num_items & 0x000000ff) << 24);

  std::vector<int> labels(num_items);
  for (int i = 0; i < num_items; ++i) {
    unsigned char label = 0;
    file.read((char *)&label, sizeof(label));
    labels[i] = static_cast<int>(label);
  }

  return labels;
}

// 读取模型参数
std::vector<float> read_param(const std::string &path) {
  std::ifstream file(path);
  std::vector<float> params;
  float param;
  while (file >> param) {
    params.push_back(param);
  }
  return params;
}

// // 范例kernel函数，无实际作用
// __global__ void add_arrays(int *a, int *b, int *c, int size) {
//   int index = threadIdx.x + blockIdx.x * blockDim.x;
//   if (index < size) {
//     c[index] = a[index] + b[index];
//   }
// }

#define ID4(i0, i1, i2, i3, d0, d1, d2, d3) (((i0) * (d1) + (i1)) * (d2) + (i2)) * (d3) + (i3)

#define ID3(i0, i1, i2, d0, d1, d2) ((i0) * (d1) + (i1)) * (d2) + (i2)

// ReLU激活函数
inline __device__ __host__ float relu(float x) { return x > 0 ? x : 0; }

// 定义卷积层的前向传播函数
void batch_conv2d_f(const float *input, const int batch_count, const int in_channels,
                    const int in_height, const int in_width, const float *weight,
                    const int out_channels, const int kernel_size, const float *bias,
                    const int stride, const int padding, float *output) {

  int out_height = (in_height - kernel_size + 2 * padding) / stride + 1;
  int out_width = (in_width - kernel_size + 2 * padding) / stride + 1;

  // 输入为 (N,C,H,W)

  // 对输入图像进行卷积操作
  // 一个 block 处理一个 (C,H,W)
  for (int b = 0; b < batch_count; b++) {

    for (int k = 0; k < out_channels; k++) {
      for (int i = 0; i < out_height; i++) {
        for (int j = 0; j < out_width; j++) {

          float sum = 0.0;
          for (int c = 0; c < in_channels; c++) {

            for (int m = 0; m < kernel_size; m++) {
              for (int n = 0; n < kernel_size; n++) {
                int input_row = i * stride + m - padding;
                int input_col = j * stride + n - padding;

                if (input_row >= 0 && input_row < in_height && input_col >= 0 &&
                    input_col < in_width) {
                  sum +=
                      (input[ID4(b, c, input_row, input_col, batch_count, in_channels, in_height,
                                 in_width)] -
                       0.5) /
                      0.5 *
                      weight[ID4(k, c, m, n, out_channels, in_channels, kernel_size, kernel_size)];
                }
              }
            }
          }
          output[ID4(b, k, i, j, batch_count, out_channels, out_height, out_width)] =
              relu(sum + bias[k]);
        }
      }
    }
  }
}

template <int KerSize, int stride, int padding, int in_channels, int in_height, int in_width,
          int out_channels>
__global__ void batch_conv2d_f_ker(float *input, const int batch_count, const int _in_channels,
                                   const int _in_height, const int _in_width, const float *weight,
                                   const int _out_channels, const int kernel_size,
                                   const float *bias, const int _stride, const int _padding,
                                   float *output, const int en_norm) {

  const int out_height = (in_height - KerSize + 2 * padding) / stride + 1;
  const int out_width = (in_width - KerSize + 2 * padding) / stride + 1;

  const int b = blockIdx.x * 4;
  const int h = threadIdx.x / out_width;
  const int w = threadIdx.x % out_width;

  // __shared__ float input_share[in_channels * in_height * in_width];
  // for (int i = threadIdx.x; i < in_channels * in_height * in_width; i += blockDim.x) {
  //   input_share[i] = input[b * in_channels * in_height * in_width + i];
  // }
  // __syncthreads();

  static_assert(out_channels == 1);
  static_assert(in_channels == 1);
  static_assert(padding == 0);

  float weight_reg[KerSize][KerSize];
  for (int m = 0; m < KerSize; m++) {
    for (int n = 0; n < KerSize; n++) {
      weight_reg[m][n] = weight[ID4(0, 0, m, n, out_channels, in_channels, KerSize, KerSize)];
    }
  }

  const int b_tile = 4;

  for (int k = 0; k < out_channels; k++) {

    // for (int h = 0; i < out_height; i++) {
    //   for (int j = 0; j < out_width; j++) {

    float sum[b_tile] = {0.0};
    for (int c = 0; c < in_channels; c++) {

#pragma unroll
      for (int bb = 0; bb < b_tile; bb++)
#pragma unroll
        for (int m = 0; m < KerSize; m++) {
#pragma unroll
          for (int n = 0; n < KerSize; n++) {
            // int n = 0;
            int input_row = h * stride + m - padding;
            int input_col = w * stride + n - padding;

            sum[bb] += (input[(b + bb) * in_channels * in_height * in_width +
                              input_row * in_weight + input_col]) *
                       weight_reg[m][n];
          }
        }
    }
    for (int bb = 0; bb < b_tile; bb++)
      output[ID4(b + bb, k, h, w, batch_count, out_channels, out_height, out_width)] =
          relu(sum[bb] + bias[k]);
    //   }
    // }
  }
  // }
}
// 线性层（Linear）
template <int InSize, int OutSize>
void batch_linear(float *input, float *weight, float *bias, float *output, const int batch_count,
                  const int en_relu) {
  for (int b = 0; b < batch_count; b++) {
    for (int i = 0; i < OutSize; i++) {
      float sum = 0.0;
      for (int j = 0; j < InSize; j++) {
        sum += input[InSize * b + j] * weight[i * InSize + j];
      }
      output[OutSize * b + i] = sum + bias[i];
      if (en_relu)
        output[OutSize * b + i] = relu(output[OutSize * b + i]);
    }
  }
}

void gemmNT_T(const float *A, const float *B, float *C, const float *bias, const int M, const int N,
              const int K, const int en_relu) {

    int mc = M, nc = N, kc = K;

    float *AC = (float *)malloc(sizeof(float) * mc * kc);
    float *BC = (float *)malloc(sizeof(float) * nc * kc);
//   for (int m = 0; m < M; m++) {   // OutSize
//     for (int n = 0; n < N; n++) { // batch_count
//       float sum = 0.0;
//       for (int k = 0; k < K; k++) { // InSize
//         sum += A[m * K + k] * B[n * K + k];
//       }
//       C[n * M + m] = sum;
//       //C[m * N + n] = sum;
//     }
//   }
    float thread_f = 3;
    const int digit = 8;

    xcopy<float>(A,AC, mc, kc);
     xcopy<float>(B,BC, kc, nc);

//xgemm_nt<float> (AC, BC, C,  mc, kc, kc, nc) ;
 xigemm_int8<float,digit>(AC, BC,C,mc,kc,kc,nc,thread_f,'x');
  for (int m = 0; m < M; m++) {   // OutSize
    for (int n = 0; n < N; n++) { // batch_count

      C[n * M + m] += bias[m];
    }
  }

  for (int m = 0; m < M; m++) {   // OutSize
    for (int n = 0; n < N; n++) { // batch_count
      if (en_relu)
        C[n * M + m] = relu(C[n * M + m]);
    }
  }
}
/**
    const int batch_tile = 8;
    const int m_tile = 2;
    gemmNT_T_ker<16 * m_tile, 16 * batch_tile, m_tile, batch_tile>
*/

template <int BLOCKX, int BLOCKY, int m_tile, int n_tile>
__global__ void gemmNT_T_ker(const float *A, const float *B, float *C, const float *bias,
                             const int M, const int N, const int K, const int en_relu) {
  const int ml = blockIdx.x * BLOCKX + threadIdx.x * m_tile;
  const int nl = blockIdx.y * BLOCKY + threadIdx.y * n_tile;

  // const int m = threadIdx.x;
  // const int n = blockIdx.x;

  const int BLOCK_K = 32;
  __shared__ float As[BLOCKX][BLOCK_K + 1];
  __shared__ float Bs[BLOCKY][BLOCK_K + 1];

  // for (int m = 0; m < M; m++) {   // OutSize
  //   for (int n = 0; n < N; n++) { // batch_count

  const int lid = threadIdx.y * (BLOCKX / m_tile) + threadIdx.x;
  const int lsize = (BLOCKX / m_tile) * (BLOCKY / n_tile);

  int kmax = (K / BLOCK_K) * BLOCK_K;

  float sum[m_tile][n_tile] = {0.0};
  for (int bkIdx = 0; bkIdx < kmax; bkIdx += BLOCK_K) {
    __syncthreads();
    for (int i = lid; i < BLOCKX * BLOCK_K; i += lsize) {
      int m = blockIdx.x * BLOCKX + i / BLOCK_K;
      int k = bkIdx + i % BLOCK_K;
      // if(m < M)
      As[i / BLOCK_K][i % BLOCK_K] = A[m * K + k];
      // else
      //   As[i / BLOCK_K][i % BLOCK_K] = 0;
    }
    for (int i = lid; i < BLOCKY * BLOCK_K; i += lsize) {
      int n = blockIdx.y * BLOCKY + i / BLOCK_K;
      int k = bkIdx + i % BLOCK_K;
      if (n < N)
        Bs[i / BLOCK_K][i % BLOCK_K] = B[n * K + k];
      else
        Bs[i / BLOCK_K][i % BLOCK_K] = 0;
    }
    __syncthreads();

    for (int k = 0; k < BLOCK_K; k++) { // InSize
      // float A_reg[2];
      // #pragma unroll
      // for(int mm = 0; mm<m_tile; mm++)
      //   A_reg[mm] = As[threadIdx.x * m_tile + mm][k];

      // #pragma unroll
      for (int nn = 0; nn < n_tile; nn++) {
        float B_reg = Bs[threadIdx.y * n_tile + nn][k];
        // #pragma unroll
        for (int mm = 0; mm < m_tile; mm++) {
          sum[mm][nn] += As[threadIdx.x * m_tile + mm][k] * B_reg;
          // sum[mm][nn] += A_reg[mm] * B_reg;
        }
      }
    }
  }
  // C[n * M + m] = sum;

  for (int nn = 0; nn < n_tile; nn++) {
    int n = nl + nn;
    for (int mm = 0; mm < m_tile; mm++) {
      int m = ml + mm;
      if (m < M && n < N) {
        // float sum = 0.0;
        for (int k = kmax; k < K; k++) { // InSize
          sum[mm][nn] += A[m * K + k] * B[n * K + k];
        }
        C[n * M + m] = sum[mm][nn] + bias[m];
        if (en_relu)
          C[n * M + m] = relu(C[n * M + m]);
      }
    }
  }
}

#define WIDTH 28
#define HEIGHT 28

void normalize(float *data, int size, float mean, float std) {
  for (int i = 0; i < size; i++) {
    data[i] = (data[i] - mean) / std;
  }
}

__global__ void normalize_ker(float *data, int size, float mean, float std) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < size)
    data[i] = (data[i] - mean) / std;
}

template <int ITEMS_PRE_THREAD, int BLOCK_SIZE>
__global__ void vec_max_index_ker(float *data, float *out_data, const int size) {
  const int common_offset = (blockIdx.x * blockDim.x) * ITEMS_PRE_THREAD;
  float max_item = data[common_offset + ITEMS_PRE_THREAD * threadIdx.x];
  int id = 0;
  __shared__ float data_shr[ITEMS_PRE_THREAD * BLOCK_SIZE];
  for(int i=0; i<ITEMS_PRE_THREAD; i++){
    data_shr[threadIdx.x + i * BLOCK_SIZE] = data[common_offset + threadIdx.x + i * BLOCK_SIZE];
  }
  __syncthreads();
  for (int i = 1; i < ITEMS_PRE_THREAD; i++) {
    // 这里可能有分支分歧但不管了，希望编译器能优化掉
    float cmp = data_shr[ITEMS_PRE_THREAD * threadIdx.x + i];
    if (cmp > max_item) {
      max_item = cmp;
      id = i;
    }
  }

  if (blockIdx.x * blockDim.x + threadIdx.x < size)
    out_data[blockIdx.x * blockDim.x + threadIdx.x] = id;
}

void output_tensor(float *data, std::vector<int> shape) {
  int dim = shape.size();

  std::vector<int> off(dim, 0);
  int size = 1;
  for (int i = 0; i < dim; i++)
    size *= shape[i];

  for (int i = 0; i < size; i++) {
    if (off[dim - 1] == 0) {
      for (int i = 0; i < dim; i++)
        printf("%3d, ", off[i]);
      printf("[");
    }
    printf("%+7.4f, ", data[i]);
    off[dim - 1]++;

    if (off[dim - 1] >= shape[dim - 1])
      printf("], \n");

    int ii = dim - 1;
    while (off[ii] >= shape[ii]) {
      off[ii] = 0;
      ii--;
      if (ii < 0)
        break; // end
      off[ii]++;
    }
  }
  printf("\n");
}

#define MIN(x, y) ((x) < (y) ? (x) : (y))

int main(int argc, char *argv[]) {
  std::string dir = argv[1];
  //     第一个参数是程序所在的目录，这个目录是存放前一步训练模型参数文件的目录，从这个目录下读取模型参数文件，相对于这个目录读取测试集图片和标签
  //          // cout << dir;

  // std::string dir = "./";
  // 读取测试集，对于想实现CUDA
  // C/C++训练的同学，参考训练集文件名为train-images-idx3-ubyte和train-labels-idx1-ubyte
  auto images = read_mnist_images(dir + "/data/FashionMNIST/raw/t10k-images-idx3-ubyte");
  float *images_data = (float *)malloc(sizeof(float) * images.size() * images[0].size());
  for (size_t i = 0; i < images.size(); i++) {
    memcpy(images_data + i * images[0].size(), images[i].data(), images[0].size() * sizeof(float));
  }

  // 读取测试集标签
  auto labels = read_mnist_labels(dir + "/data/FashionMNIST/raw/t10k-labels-idx1-ubyte");
  // 读取模型参数
  auto conv1_weight = read_param(dir + "/conv1.weight.txt");
  auto conv1_bias = read_param(dir + "/conv1.bias.txt");
  auto conv2_weight = read_param(dir + "/conv2.weight.txt");
  auto conv2_bias = read_param(dir + "/conv2.bias.txt");
  auto fc1_weight = read_param(dir + "/fc1.weight.txt");
  auto fc1_bias = read_param(dir + "/fc1.bias.txt");
  auto fc2_weight = read_param(dir + "/fc2.weight.txt");
  auto fc2_bias = read_param(dir + "/fc2.bias.txt");
  // auto fc3_weight = read_param(dir + "/fc3.weight.txt");
  // auto fc3_bias = read_param(dir + "/fc3.bias.txt");

  float *conv1_weight_dev;
  float *conv1_bias_dev;
  float *conv2_weight_dev;
  float *conv2_bias_dev;
  float *fc1_weight_dev;
  float *fc1_bias_dev;
  float *fc2_weight_dev;
  float *fc2_bias_dev;

  checkCudaErrors(hipMalloc((void **)&conv1_weight_dev, sizeof(float) * conv1_weight.size()));
  checkCudaErrors(hipMalloc((void **)&conv1_bias_dev, sizeof(float) * conv1_bias.size()));
  checkCudaErrors(hipMalloc((void **)&conv2_weight_dev, sizeof(float) * conv2_weight.size()));
  checkCudaErrors(hipMalloc((void **)&conv2_bias_dev, sizeof(float) * conv2_bias.size()));
  checkCudaErrors(hipMalloc((void **)&fc1_weight_dev, sizeof(float) * fc1_weight.size()));
  checkCudaErrors(hipMalloc((void **)&fc1_bias_dev, sizeof(float) * fc1_bias.size()));
  checkCudaErrors(hipMalloc((void **)&fc2_weight_dev, sizeof(float) * fc2_weight.size()));
  checkCudaErrors(hipMalloc((void **)&fc2_bias_dev, sizeof(float) * fc2_bias.size()));

  checkCudaErrors(hipMemcpy(conv1_weight_dev, conv1_weight.data(),
                             sizeof(float) * conv1_weight.size(), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(conv1_bias_dev, conv1_bias.data(), sizeof(float) * conv1_bias.size(),
                             hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(conv2_weight_dev, conv2_weight.data(),
                             sizeof(float) * conv2_weight.size(), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(conv2_bias_dev, conv2_bias.data(), sizeof(float) * conv2_bias.size(),
                             hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(fc1_weight_dev, fc1_weight.data(), sizeof(float) * fc1_weight.size(),
                             hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(fc1_bias_dev, fc1_bias.data(), sizeof(float) * fc1_bias.size(),
                             hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(fc2_weight_dev, fc2_weight.data(), sizeof(float) * fc2_weight.size(),
                             hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(fc2_bias_dev, fc2_bias.data(), sizeof(float) * fc2_bias.size(),
                             hipMemcpyHostToDevice));

  int correct = 0;
  std::vector<float> pred_labels(labels.size(), -1);

  const int conv_1_out_size = (conv_1_out_channel * conv_1_out_height * conv_1_out_width);

  float *conv_1_out = (float *)malloc(sizeof(float) * images.size() * conv_1_out_size);
  float *linear_1_out = (float *)malloc(sizeof(float) * images.size() * linear_1_out_size);
  float *linear_2_out= (float *)malloc(sizeof(float) * images.size() * linear_2_out_size);




  float *conv_1_out_dev;
  float *linear_1_out_dev;
  float *linear_2_out_dev;

  checkCudaErrors(
      hipMalloc((void **)&conv_1_out_dev, sizeof(float) * batch_size * conv_1_out_size));
  checkCudaErrors(
      hipMalloc((void **)&linear_1_out_dev, sizeof(float) * images.size() * linear_1_out_size));
  checkCudaErrors(hipMalloc((void **)&linear_2_out_dev,
                             sizeof(float) * images.size() * linear_2_out_size)); // 完整大小

  float *images_data_dev_ext;
  checkCudaErrors(hipMalloc((void **)&images_data_dev_ext,
                             sizeof(float) * (images.size() + 2) * (images[0].size() + 2)));
  float *images_data_dev = images_data_dev_ext + (images[0].size() + 2) * 2 + 2;
  checkCudaErrors(hipMemcpy(images_data_dev, images_data,
                             sizeof(float) * images.size() * images[0].size(),
                             hipMemcpyHostToDevice));

  // 开始计时，使用chrono计时，不支持其它计时方式
  auto start = std::chrono::high_resolution_clock::now();
  timeprof_start_("all");

  for (size_t batch_off = 0; batch_off < images.size(); batch_off += batch_size) {
    int batch_count = MIN(batch_off + batch_size, images.size()) - batch_off;


    {
        //0.000151
      batch_conv2d_f_ker<2, conv_1_stride, conv_1_padding, in_channel, in_height, in_weight,
                         conv_1_out_channel>
          <<<batch_count / 4, conv_1_out_height * conv_1_out_width>>>(
              images_data_dev + images[0].size() * batch_off, batch_count, in_channel, in_height,
              in_weight, conv1_weight_dev, conv_1_out_channel, conv_1_kernel_size, conv1_bias_dev,
              conv_1_stride, conv_1_padding, conv_1_out_dev, 1);
    }

    checkCudaErrors(hipGetLastError());
    const int batch_tile = 8;
    const int m_tile = 2;


    //0.000174
    {
      dim3 gridDim2(CEIL_DIV(linear_1_out_size, 16 * m_tile), CEIL_DIV(batch_count, 16 * batch_tile));
      dim3 blockDim2(16, 16);
      //0.000055


        checkCudaErrors(hipMemcpy(conv_1_out, conv_1_out_dev, sizeof(float) * batch_size * conv_1_out_size,
                              hipMemcpyDeviceToHost));

        // xgemm_nt<float>(fc1_weight.data(), conv_1_out, linear_1_out, fc1_bias.data(),
        //                             linear_1_out_size, batch_count,batch_count, linear_1_in_size);

       
        gemmNT_T(fc1_weight.data(), conv_1_out, linear_1_out, fc1_bias.data(),
                                    linear_1_out_size, batch_count, linear_1_in_size, 1);

        checkCudaErrors(hipMemcpy(linear_1_out_dev, linear_1_out,
                                    sizeof(float) * images.size() * linear_1_out_size,
                                    hipMemcpyHostToDevice));

    //   gemmNT_T_ker<16 * m_tile, 16 * batch_tile, m_tile, batch_tile>
    //       <<<gridDim2, blockDim2>>>(fc1_weight_dev, conv_1_out_dev, linear_1_out_dev, fc1_bias_dev,
    //                                 linear_1_out_size, batch_count, linear_1_in_size, 1);
    }

    checkCudaErrors(hipGetLastError());

    {
      const int batch_tile = 1;
      dim3 gridDim2(CEIL_DIV(linear_2_out_size, 10), CEIL_DIV(batch_count, 16 * batch_tile));
      dim3 blockDim2(10, 16);
      //0.000018:


      checkCudaErrors(hipMemcpy(linear_1_out, linear_1_out_dev, sizeof(float) * images.size() * linear_1_out_size,
                              hipMemcpyDeviceToHost));


      gemmNT_T(
          fc2_weight.data(), linear_1_out, linear_2_out + batch_off * linear_2_out_size,
          fc2_bias.data(), linear_2_out_size, batch_count, linear_1_out_size, 0);

    //   gemmNT_T_ker<10, 16 * batch_tile, 1, batch_tile><<<gridDim2, blockDim2>>>(
    //       fc2_weight_dev, linear_1_out_dev, linear_2_out_dev + batch_off * linear_2_out_size,
    //       fc2_bias_dev, linear_2_out_size, batch_count, linear_1_out_size, 0);

        checkCudaErrors(hipMemcpy(linear_2_out_dev, linear_2_out,
                                    sizeof(float) * images.size() * linear_2_out_size,
                                    hipMemcpyHostToDevice));
      
    }

    checkCudaErrors(hipGetLastError());


    vec_max_index_ker<10, 128>
        <<<CEIL_DIV(batch_count, 128), 128>>>(linear_2_out_dev + batch_off * linear_2_out_size,
                                              linear_2_out_dev + batch_off, batch_count);
    checkCudaErrors(hipGetLastError());

  }


//     //0.000150
  checkCudaErrors(hipMemcpy(linear_2_out, linear_2_out_dev, sizeof(float) * images.size(),
                             hipMemcpyDeviceToHost));


//   // 向主机端同步以等待所有异步调用的GPU kernel执行完毕，这句必须要有
  hipDeviceSynchronize();

  // 结束计时
  auto end = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> diff = end - start;
  timeprof_end_();

  for (size_t batch_off = 0; batch_off < images.size(); batch_off += batch_size) {
    for (size_t t = batch_off; t < MIN(batch_off + batch_size, images.size()); t++) {
      int pred_label = linear_2_out[t];
      // printf("pred_label=%d\n", pred_label);
      if (pred_label == labels[t]) {
        correct++;
      }
    }
  }
  timeprof_end_();

  // 输出结果，请严格保持此输出格式，并把0.0001替换成实际的准确率，请不要输出除了此结果之外的任何内容！！！
  std::cout << std::fixed << std::setprecision(6) << diff.count() << ":" << std::setprecision(4)
            << (double)correct / labels.size() << std::endl;

  timeprof_print_frame_sorted_();

  return 0;
}